#include "../include/DataLoader.cuh"
DataLoader::DataLoader(const std::string& data_path, const int di, bool genXW):dim(di){
    cpuA = std::make_unique<CSR>();
    std::fstream fin;
    fin.open(data_path,std::ios::in);
    //std::cout<<this->data_path<<std::endl;
    //std::cout<<name0<<std::endl;
    //std::cout<<this->data_path+"\/"+"n_"+name0+".csv"<<std::endl;
    std::string line, word;
    
    std::getline(fin,line);
    std::stringstream ss1(line);
    while(std::getline(ss1,word,',')){
        cpuA->row.push_back(std::stoi(word));        
    }
    
    std::getline(fin,line);
    std::stringstream ss2(line);
    while(std::getline(ss2,word,',')){
        cpuA->col.push_back(std::stoi(word));        
    }

    std::getline(fin,line);
    std::stringstream ss3(line);
    while(std::getline(ss3,word,',')){
        cpuA->vals.push_back(std::stof(word));        
    }
    assert(cpuA->col.size()==cpuA->vals.size());
    n = cpuA->row.size()-1; 
    cpuA->r = cpuA->row.size()-1; 
    cpuA->c = cpuA->row.size()-1; 
    cpuA->nnz = cpuA->col.size();
    fin.close(); 

    std::string data_name = data_path.substr(data_path.find_last_of("/")+1,-1);
    graph_name = data_name.substr(0, data_name.find(".")); 
    if (data_name == "polblogs.csv"){
        c = 2; 
    }else if(data_name == "cora.csv"){
        c = 7; 
    }else if (data_name == "citeseer.csv"){
        c = 6; 
    }else if (data_name == "pubmed.csv"){
        c = 3; 
    }else if (data_name == "ppi.csv"){
        c = 121; 
    }else if (data_name == "reddit.csv"){
        c = 41; 
    }else if (data_name == "flickr.csv"){
        c = 7; 
    }else if (data_name == "yelp.csv"){
        c = 100; 
    }else if (data_name == "amazon.csv"){
        c = 107; 
    }else{
        std::cout<<"not supported data"<<std::endl;
        exit(0);
    }
    gpuA = std::make_unique<dCSR>();
    if (genXW){
        if (alloc()){
            LOG(INFO) << "Initialize X & W ...";
            for (int i=0; i<n*dim; ++i){
                cpuX[i] = rand()/RAND_MAX;
            }
            for (int i=0; i<c*dim; ++i){
                cpuW[i] = rand()/RAND_MAX;
            }
            LOG(INFO) << "X & W initialized ...";
            //print_data();
            transfer();
        } 
    }
}

bool DataLoader::transfer(){
    LOG(INFO) << "Transfer A, X & W to gpu ...";
    CUDA_CHECK(hipMemcpy(gpuA->row, cpuA->row.data(), sizeof(unsigned int)*(cpuA->r+1), hipMemcpyHostToDevice));
    LOG(INFO) << "Transfer A row ...";
    CUDA_CHECK(hipMemcpy(gpuA->col, cpuA->col.data(), sizeof(unsigned int)*cpuA->nnz, hipMemcpyHostToDevice));
    LOG(INFO) << "Transfer A, col ...";
    CUDA_CHECK(hipMemcpy(gpuA->vals, cpuA->vals.data(), sizeof(T)*cpuA->nnz, hipMemcpyHostToDevice));
    
    LOG(INFO) << "Transfer A, X & W to gpu ...";
    CUDA_CHECK(hipMemcpy(gpuX, &cpuX[0], sizeof(T)*n*dim, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(gpuW, &cpuW[0], sizeof(T)*dim*c, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(gpuC, 0, sizeof(T)*n*c));
    LOG(INFO) << "A, X & W have transfered to gpu ...";
    return true;
}

bool DataLoader::alloc(){
    cpuX = std::make_unique<T []>(n*dim);
    cpuW = std::make_unique<T []>(c*dim);
    cpuC = std::make_unique<T []>(n*c);
    //memset(cpuC, 0, sizeof(T)*n*c);

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&(gpuA->row)), sizeof(unsigned int) * (cpuA->r+1)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&(gpuA->col)), sizeof(unsigned int) * (cpuA->nnz)));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&(gpuA->vals)), sizeof(T) * (cpuA->nnz)));
    
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&gpuX), sizeof(T) * n * dim));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&gpuW), sizeof(T) * c * dim));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&gpuC), sizeof(T) * c * n));
    return true;
}

void DataLoader::print_data(){
    LOG(INFO) << "print start.";
    std::cout<<"The first 5 elements of rowptr: ";
    for(auto it=cpuA->row.begin(); it<cpuA->row.begin()+5; it++)
        std::cout<<(*it)<<" ";
    std::cout<<std::endl;

    std::cout<<"The last 5 elements of rowptr: ";
    for(auto it=cpuA->row.end()-5; it!=cpuA->row.end() ; it++)
        std::cout<<(*it)<<" ";
    std::cout<<std::endl;

    std::cout<<"The first 5 elements of indies: ";
    for(auto it=cpuA->col.begin(); it<cpuA->col.begin()+5 ; it++)
        std::cout<<(*it)<<" ";
    std::cout<<std::endl;

    std::cout<<"The last 5 elements of indies: ";
    for(auto it=cpuA->col.end()-5; it!=cpuA->col.end() ; it++)
        std::cout<<(*it)<<" ";
    std::cout<<std::endl;

    std::cout<<"The first 5 elements of vals: ";
    for(auto it=cpuA->vals.begin(); it<cpuA->vals.begin()+5 ; it++)
        std::cout<<(*it)<<" ";
    std::cout<<std::endl;

    std::cout<<"The last 5 elements of vals: ";
    for(auto it=cpuA->vals.end()-5; it!=cpuA->vals.end() ; it++)
        std::cout<<(*it)<<" ";
    std::cout<<std::endl;
    
    std::cout<<"The first 5 elements of X: ";
    for(auto it=0; it<5 ; it++)
        std::cout<<cpuX[it]<<" ";
    std::cout<<std::endl;
    
    std::cout<<"The first 5 elements of W: ";
    for(auto it=0; it<5 ; it++)
        std::cout<<cpuW[it]<<" ";
    std::cout<<std::endl;
    
    std::cout<<std::endl;
    //std::cout<<"The number of nodes: "<< get_nodes()<<"   Rowptr: "<<data.at(0).size()<<"   Pointer: "<<data.at(1).size()<<std::endl;
    //std::cout<<"The size of a node feature: "<<get_feature_size()<<std::endl;
}
