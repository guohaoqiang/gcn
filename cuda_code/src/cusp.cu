#include "../include/cusp.cuh"

int run1(DataLoader& input, Metrics& metric){
    T *gpuB = nullptr; // n * c
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&gpuB), sizeof(T) * input.n * input.c));
    float duration;
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    // ############################
    hipEventRecord(start);
    // ############################

    //----------  B = XW : sgemm------------
    const float alpha = 1.0;
    const float beta = 0.0;
    
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: compute */
    CUBLAS_CHECK(hipblasSgemm(cublasH, transa, transb, input.n, input.c, input.dim, &alpha, 
                input.gpuX, input.n, input.gpuW, input.dim, &beta, gpuB, input.n));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    LOG(INFO) << "step1 of run1 completed ...";

    //----------  C = AB : sparsemm------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, input.cpuA->r, input.cpuA->c, input.cpuA->nnz,
                                      input.gpuA->row, input.gpuA->col, input.gpuA->vals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, input.n, input.c, input.n, gpuB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, input.n, input.c, input.n, input.gpuRef1,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )

    LOG(INFO) << "step2 of run1 completed ...";
    // ############################
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	// ############################
	hipEventElapsedTime(&duration, start, stop);
    metric.t += duration;
    metric.flops = (input.cpuA->nnz * input.c + input.n * input.dim * input.c) * 2;
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    CHECK_CUDA( hipFree(dBuffer) )
    CUDA_CHECK( hipFree(gpuB) );

    CUDA_CHECK(hipMemcpy(&(input.cpuRef1[0]), input.gpuRef1, sizeof(T)*input.n*input.c, hipMemcpyDeviceToHost));
    LOG(INFO) << "run1 completed ...";
    return 0;
}

int run2(DataLoader& input, Metrics& metric){
    T *gpuB = nullptr; // n * dim
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&gpuB), sizeof(T) * input.n * input.dim));
    float duration;
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    // ############################
    hipEventRecord(start);
    // ############################

    //----------  B = AX : sparsemm------------
    const float alpha = 1.0;
    const float beta = 0.0;
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, input.cpuA->r, input.cpuA->c, input.cpuA->nnz,
                                      input.gpuA->row, input.gpuA->col, input.gpuA->vals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix X
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, input.n, input.dim, input.n, input.gpuX,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, input.n, input.dim, input.n, gpuB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer) )
    LOG(INFO) << "step1 of run2 completed ...";
    //----------  C = BW : sgemm------------
    
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: compute */
    CUBLAS_CHECK(hipblasSgemm(cublasH, transa, transb, input.n, input.c, input.dim, &alpha, 
                gpuB, input.n, input.gpuW, input.dim, &beta, input.gpuRef2, input.n));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    LOG(INFO) << "step2 of run2 completed ...";

    // ############################
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	// ############################
	hipEventElapsedTime(&duration, start, stop);
    metric.t += duration;
    metric.flops = (input.cpuA->nnz * input.dim + input.n * input.dim * input.c) * 2;
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    CHECK_CUDA( hipFree(dBuffer) )
    CUDA_CHECK( hipFree(gpuB) );

    CUDA_CHECK(hipMemcpy(&(input.cpuRef2[0]), input.gpuRef2, sizeof(T)*input.n*input.c, hipMemcpyDeviceToHost));
    LOG(INFO) << "run2 completed ...";
    return 0;
}

