#include "../include/cusp.cuh"

int run1(DataLoader& input, Metrics& metric){
    T *gpuB = nullptr; // n * c
    CUDA_CHECK(hipMalloc(&gpuB, sizeof(T) * input.n * input.c));
    float duration, spgemm_duration, gemm_duration;
    hipEvent_t start, stop, spgemm_start, spgemm_stop, gemm_start, gemm_stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&spgemm_start);
	hipEventCreate(&spgemm_stop);
	hipEventCreate(&gemm_start);
	hipEventCreate(&gemm_stop);
    // ############################
    hipEventRecord(start);
    hipEventRecord(gemm_start);
    // ############################

    //----------  B = XW : sgemm------------
    const float alpha = 1.0;
    const float beta = 0.0;
    
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: compute */
    CUBLAS_CHECK(hipblasSgemm(cublasH, transa, transb, input.n, input.c, input.dim, &alpha, 
                input.gpuX, input.n, input.gpuW, input.dim, &beta, gpuB, input.n));
    CUBLAS_CHECK(hipblasDestroy(cublasH));
    //LOG(INFO) << "step1 of run1 completed ...";
	hipEventRecord(gemm_stop);
	hipEventSynchronize(gemm_stop);

    //----------  C = AB : sparsemm------------
    hipEventRecord(spgemm_start);
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, input.cpuA->r, input.cpuA->c, input.cpuA->nnz,
                                      input.gpuA->row, input.gpuA->col, input.gpuA->vals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, input.n, input.c, input.n, gpuB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix C
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, input.n, input.c, input.n, input.gpuRef1,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG3, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG3, dBuffer) )
	hipEventRecord(spgemm_stop);
	hipEventSynchronize(spgemm_stop);

    //LOG(INFO) << "step2 of run1 completed ...";
    // ############################
	//hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	// ############################
	hipEventElapsedTime(&duration, start, stop);
	hipEventElapsedTime(&gemm_duration, gemm_start, gemm_stop);
	hipEventElapsedTime(&spgemm_duration, spgemm_start, spgemm_stop);
    metric.t += duration;
    metric.spgemm_t += spgemm_duration;
    metric.gemm_t += gemm_duration;
    metric.flops = (input.cpuA->nnz * input.c + input.n * input.dim * input.c) * 2;
    metric.spgemm_flops = (input.cpuA->nnz * input.c) * 2;
    metric.gemm_flops = (input.n * input.dim * input.c) * 2;
    //                              A                  X                 W                    B
    metric.dataMovement = 4*(input.cpuA->nnz + input.n*input.dim + input.dim*input.c + 2*input.n*input.c);
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    CHECK_CUDA( hipFree(dBuffer) )
    CUDA_CHECK( hipFree(gpuB) );

    CUDA_CHECK(hipMemcpy(&(input.cpuRef1[0]), input.gpuRef1, sizeof(T)*input.n*input.c, hipMemcpyDeviceToHost));
    LOG(INFO) << "run1 completed ...";
    return 0;
}

int run2(DataLoader& input, Metrics& metric){
    T *gpuB = nullptr; // n * dim
    CUDA_CHECK(hipMalloc(&gpuB, sizeof(T) * input.n * input.dim));
    float duration, spgemm_duration, gemm_duration;
    hipEvent_t start, stop, spgemm_start, spgemm_stop, gemm_start, gemm_stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&spgemm_start);
	hipEventCreate(&spgemm_stop);
	hipEventCreate(&gemm_start);
	hipEventCreate(&gemm_stop);
    // ############################
    hipEventRecord(start);
    // ############################

    //----------  B = AX : sparsemm------------
    hipEventRecord(spgemm_start);
    const float alpha = 1.0;
    const float beta = 0.0;
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, input.cpuA->r, input.cpuA->c, input.cpuA->nnz,
                                      input.gpuA->row, input.gpuA->col, input.gpuA->vals,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense matrix X
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, input.n, input.dim, input.n, input.gpuX,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matC, input.n, input.dim, input.n, gpuB,
                                        HIP_R_32F, HIPSPARSE_ORDER_COL) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMM_bufferSize(
                                 handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG3, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMM
    CHECK_CUSPARSE( hipsparseSpMM(handle,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                 HIPSPARSE_SPMM_CSR_ALG3, dBuffer) )
    //LOG(INFO) << "step1 of run2 completed ...";
	hipEventRecord(spgemm_stop);
	hipEventSynchronize(spgemm_stop);
    //----------  C = BW : sgemm------------
    
    hipEventRecord(gemm_start);
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    /* step 1: create cublas handle, bind a stream */
    hipblasHandle_t cublasH = NULL;
    CUBLAS_CHECK(hipblasCreate(&cublasH));

    /* step 2: compute */
    CUBLAS_CHECK(hipblasSgemm(cublasH, transa, transb, input.n, input.c, input.dim, &alpha, 
                gpuB, input.n, input.gpuW, input.dim, &beta, input.gpuRef2, input.n));
    CUBLAS_CHECK(hipblasDestroy(cublasH));

    //LOG(INFO) << "step2 of run2 completed ...";
	hipEventRecord(gemm_stop);
	hipEventSynchronize(gemm_stop);

    // ############################
	//hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	// ############################
	hipEventElapsedTime(&duration, start, stop);
	hipEventElapsedTime(&spgemm_duration, spgemm_start, spgemm_stop);
	hipEventElapsedTime(&gemm_duration, gemm_start, gemm_stop);
    metric.t += duration;
    metric.spgemm_t += spgemm_duration;
    metric.gemm_t += gemm_duration;
    metric.flops = (input.cpuA->nnz * input.dim + input.n * input.dim * input.c) * 2;
    metric.spgemm_flops = (input.cpuA->nnz * input.dim) * 2;
    metric.gemm_flops = (input.n * input.dim * input.c) * 2;
    //                              A                  X                 W                    B
    metric.dataMovement = 4*(input.cpuA->nnz + input.n*input.dim + input.dim*input.c + 2*input.n*input.dim);
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    CHECK_CUDA( hipFree(dBuffer) )
    CUDA_CHECK( hipFree(gpuB) );

    CUDA_CHECK(hipMemcpy(&(input.cpuRef2[0]), input.gpuRef2, sizeof(T)*input.n*input.c, hipMemcpyDeviceToHost));
    LOG(INFO) << "run2 completed ...";
    return 0;
}

